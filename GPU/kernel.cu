#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "..\usr\include\GL\freeglut.h"
#include <stdio.h>
#include <time.h>
#include <math.h>



//�ݹ� �Լ�
void Render();
void Reshape(int w, int h);
void Timer(int id);

//����� ���� �Լ�
void CreateTree();
void Treedraw(float* location, int start);
__global__ void TreeKernel(float* dev_location, float* dev_angle, float len, int find, float angular);


#define TILE_WIDTH 32
const int Dim = 1024;
float* location;
float* angle;
float len = 0.15;
int num = 1;
int start = 0;
float angular = 0.0f;

int main(int argc, char** argv)
{
	printf("����:");
	scanf_s("%d", &num);
	printf("����:");
	scanf_s("%f", &angular);
	location = (float*)malloc(sizeof(float) * 2 * 3);
	angle = (float*)malloc(sizeof(float) * 3);
	for (int i = 0; i < 6; i += 2)
	{
		location[i] = 0.0f;
		location[i + 1] = -0.5f;
	}
	for (int i = 0; i < 3; i++)
	{
		angle[i] = 0;
	}

	//GLUT �ʱ�ȭ
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB);

	//������ ũ�� ���� �� ����
	glutInitWindowSize(Dim, Dim);
	glutCreateWindow("Fractal Tree(GPU)");

	//�ݹ� �Լ� ���
	glutDisplayFunc(Render);
	glutReshapeFunc(Reshape);
	glutTimerFunc(300, Timer, 0);

	hipSetDevice(0);

	//�̺�Ʈ ó�� ���� ����
	glutMainLoop();

	hipDeviceReset();
	return 0;
}

void Treedraw(float* location, int start)
{
	if (start == 1)
	{
		glColor3f(0.58, (float)1 / num, 0);
		glLineWidth(3.0f);
		glBegin(GL_LINES);
		glVertex2f(location[0], location[1]);
		glVertex2f(0.0, -1.0);
	}

	int k = 0;
	int n = 0;
	float g = (float)1 / num;
	float r = (float)0.58 / num;
	for (int i = 0; i < (pow(2, start - 1) - 1); i++)
	{
		// ������ ���� ���� ������ ���� �ڵ�
		if (k > pow(2, n))
		{
			g += (float)1 / num;
			r += (float)0.58 / num;
			n++;
		}
		// Left Tree
		glColor3f(0.58 - r, g, 0);
		glLineWidth(3.0f);
		glBegin(GL_LINES);
		glVertex2f(location[2 * i], location[2 * i + 1]);
		glVertex2f(location[4 * i + 2], location[4 * i + 3]);

		// Right Tree
		glColor3f(0.58 - r, g, 0);
		glLineWidth(3.0f);
		glBegin(GL_LINES);
		glVertex2f(location[2 * i], location[2 * i + 1]);
		glVertex2f(location[4 * i + 4], location[4 * i + 5]);
		k++;
	}
	glEnd();
}


void Render()
{
	if (num < start)
	{
		exit(0);
	}
	CreateTree();
	Treedraw(location, start);
	len = 0.89 * len;
	start++;
	int need = pow(2, start + 2) - 1;
	location = (float*)realloc(location, sizeof(float) * 2 * need);
	angle = (float*)realloc(angle, sizeof(float) * need);
	glFinish();
}

void Reshape(int w, int h)
{
	glViewport(0, 0, w, h);
}



void Timer(int id)
{
	glutPostRedisplay();
	glutTimerFunc(500, Timer, 0);
}

void CreateTree()
{
	// ��ǥ ���� �迭�� �ε��� ����� ���� ���� find, check
	int find = pow(2, start + 2) - 1;		// ��ü Ʈ���� �׸��µ� �ʿ��� ��ǥ�� �� ����
	int check = pow(2, start);				// ������ ������ ��ǥ�� ����

	float* dev_location;	// GPU ������ ��ǥ�� �����ϱ� ���� �迭
	float* dev_angle;		// GPU ������ ��ǥ������ ������ �����ϱ� ���� �迭

	hipError_t cudaStatus = hipSetDevice(0);
	hipMalloc((void**)&dev_location, sizeof(float) * 2 * find);
	hipMalloc((void**)&dev_angle, sizeof(float) * find);
	hipMemcpy(dev_location, location, sizeof(float) * 2 * find, hipMemcpyHostToDevice);
	hipMemcpy(dev_angle, angle, sizeof(float) * find, hipMemcpyHostToDevice);

	dim3 gridDim(128, 128);
	dim3 blockDim(TILE_WIDTH, TILE_WIDTH);

	clock_t st = clock();
	TreeKernel << <gridDim, blockDim >> > (dev_location, dev_angle, len, check, angular);
	hipDeviceSynchronize();

	hipMemcpy(location, dev_location, sizeof(float) * 2 * find, hipMemcpyDeviceToHost);
	hipMemcpy(angle, dev_angle, sizeof(float) * find, hipMemcpyDeviceToHost);

	hipFree(dev_location);
	hipFree(dev_angle);

	if (start > 0)
		printf("%d��° Elapsed time = %u ms\n", start, clock() - st);
}

__global__ void TreeKernel(float* dev_location, float* dev_angle, float len, int check, float angular)
{
	int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
	int y = blockIdx.y * TILE_WIDTH + threadIdx.y;

	int index = x + y * 4096;
	int destination = 2 * (check - 1) + 2 * index;	// ���� Ʈ���� ��ǥ ������ ���� �ε���

	// ���� index (0 ~ check-1) ���� dev_location�� ��ǥ���� ����Ǿ� ����
	if (index < check)
	{
		float a = dev_location[destination];
		float b = dev_location[destination + 1];
		float angle = dev_angle[check - 1 + index];

		// ���� Ʈ���� x,y ��ǥ
		float lx = a + len * sin(angle - angular);
		float ly = b + len * cos(angle - angular);
		dev_location[2 * destination + 2] = lx;
		dev_location[2 * destination + 3] = ly;
		dev_angle[2 * check + 2 * index - 1] = angle - angular;		// ������ Ʈ���� ���ϴ� ���� ����

		// ������ Ʈ���� x,y ��ǥ
		float rx = a + len * sin(angle + angular);
		float ry = b + len * cos(angle + angular);
		dev_location[2 * destination + 4] = rx;
		dev_location[2 * destination + 5] = ry;
		dev_angle[2 * check + 2 * index] = angle + angular;		// ���� Ʈ���� ���ϴ� ���� ����
	}
}